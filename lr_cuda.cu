#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <utility>
#include <fstream>
#include <cstring>
#include <cmath>
#ifdef _WIN32
#include "windows.h"
#endif

using namespace std;

#define WORD_SIZE 128
#define LINE_WIDTH 12800

#define TARGET_ACCURACY 0.99
#define FEATURE_DIM 30
#define BATCH_SIZE 256
#define BATCH_TOTAL 1024
#define ERROR_LAST_ITERATION 30
#define LEARNING_RATE 0.01
#define LINES_TO_READ 977
#define TRAIN_RECORD_NUM 777
#define TEST_RECORD_NUM 200


// MACRO & FUNCTIONS
#define F_SIGMOID( x ) ( 1.0 / ( 1.0 + expf( -x ) ) )

float cpu_dot(const vector< pair<int, float> > & a, const vector<float> & b) {
	float ret = 0.0;
	for (vector< pair<int, float> >::const_iterator i = a.begin();
		i != a.end(); i++) {
		ret += i->second * b[i->first];
	}
	return ret;
}

vector<float> cpu_batch_dot(const vector< vector< pair<int, float> > > & data, const vector<float> & b) {
	vector<float> rets(data.size(), 0);
	for (int i = 0; i < data.size(); i++) {
		rets[i] = cpu_dot(data[i], b);
	}
	return rets;
}

double sigmoid(float x) {
	return 1.0 / (1.0 + exp(-1.0 * x));
}

double cpu_grad(const vector< pair<int, float> > & x,
	const float wtx,
	const int label,
	vector<float> & w,
	const float learning_rate,
	const float lambda) {
	//float err = (float)label - sigmoid(wtx);
	float err = (float)label - F_SIGMOID( wtx );
	for (vector< pair<int, float> >::const_iterator i = x.begin();
		i != x.end(); i++) {
		w[i->first] += learning_rate * (err - lambda * w[i->first]);
	}
	return abs(err);
}

double cpu_batch_grad(const vector< vector< pair<int, float> > > & data,
	const vector< int > & label,
	vector<float> & b,
	const float learning_rate,
	const float lambda) {
	vector<float> dot = cpu_batch_dot(data, b);
	float err = 0.;
	float total = 0.;
	for (int i = 0; i < data.size(); i++) {
		err += cpu_grad(data[i], dot[i], label[i], b, learning_rate, lambda);
		total += 1.;
	}
	return err / total;
}


void mock_sample(const int max_feature_id, vector< pair<int, float> > & out, int * label) {
	int base = (int)floor(0.2*max_feature_id);
	int count = rand() % (1-base) + base;
	int ret = 0;
	for(int i = 0; i < count; i++) {
		int fid = rand() % max_feature_id;
		if(fid % 2 == 0) ret += 1;
		else ret -= 1;
		if(abs(ret) > 10) break;
		out.push_back(make_pair<int, float>(fid, 1.0));
	}
	*label = (ret > 0) ? 1 : 0;
}


// get data from dataset
void prepare_sample_batch( void * data, vector< pair<int, float> > & out, int * label ) {
    float * mat = (float*) data;
    // get input vector
    int i;
    int j = rand() % TRAIN_RECORD_NUM;
    for ( i=0; i<FEATURE_DIM-1; i++ ) {
        if ( mat[j * FEATURE_DIM + i] != 0 ) {
            out.push_back( make_pair<int, float>(i, mat[j * FEATURE_DIM + i] ) );
        }
    }
    // add bias
    out.push_back( make_pair<int, float>( FEATURE_DIM - 1, 1.0 ) );
    *label = (int) mat[j * FEATURE_DIM + FEATURE_DIM - 1];
}


__global__ void cuda_lr ( void * _data, float * _model ) {
	fprintf( stdout, "============== TRAIN MODEL ===============\n" );
	float learning_rate = LEARNING_RATE;
	float lambda = 0.00;
	float err;
	float err_hist[ERROR_LAST_ITERATION]; // error last for a low level
	int i, j, l, err_id = 0;
	float err_tot = 0;
	for ( i=0; i<ERROR_LAST_ITERATION; i++) {
		err_hist[i] = 1.0;
		err_tot += err_hist[i];
	}
    float * model = (float*)hipMalloc( sizeof(float) * FEATURE_DIM ); /*__device__*/
	// initialize model
	for (i = 0; i < model.size(); i++) {
        _model[i] = 0.5 - (double)(rand() % 10000) / 10000.0;
	}
    hipMemcpy( (char*)model, (char*)_model, sizeof(float)*FEATURE_DIM, hipMemcpyHostToDevice);
	// mini-batch algorithm with steepest-descent method
    // training matrix
    float * data = (float*)hipMalloc( sizeof(float) * FEATURE_DIM * TRAIN_RECORD_NUM );
    hipMemcpy(
        (char*)data,
        (char*)_data,
        sizeof(float)*FEATURE_DIM*TRAIN_RECORD_NUM,
        hipMemcpyHostToDevice
    );

	int * _seq = (int*)malloc( sizeof(int) * BATCH_SIZE * BATCH_TOTAL ); /* __host__ */
    int * seq = (int*)hipMalloc( sizeof(int) * BATCH_SIZE * BATCH_TOTAL ); /* __device__ */

	for (i = 0; i < BATCH_TOTAL; i++) {
		for (j = 0; j < ; j++) {
			_seq[] = rand()%TRAIN_RECORD_NUM;
		}
		// caculate the error
		err = cpu_batch_grad(batch_data, batch_label, model,
			learning_rate, lambda);
		//cout << "iter#" << i << " mean error: " << err << endl;

		// update error history
		err_tot -= err_hist[err_id];
		err_tot += err;
		err_hist[err_id] = err;
		err_id = (err_id+1)%ERROR_LAST_ITERATION;

		if (err_tot/ERROR_LAST_ITERATION < 1.0 - TARGET_ACCURACY) {
			std::cout << "Target accuracy achieved!\n" << std::endl;
			break;
		}
	}
	std::cout << "training accuracy: " << 1.0-err_tot/ERROR_LAST_ITERATION << std::endl;
	// print weights
	for (i = 0; i < model.size(); i++) {
		_model[i] = model[i];
		std::cout << model[i] << " ";
	}
	std::cout << std::endl;
}


// split line into vectors of words
void split_line(
        std::vector< std::string > & words,
        const char * str,
        char delim ) {
    int i, j;
    char w[WORD_SIZE];
    words.clear();
    i = j = 0;
    for( ; i<LINE_WIDTH; i++ ) {
        if ( !str[i] ) {
            w[j] = 0;
            words.push_back( std::string( w ) );
            break;
        } else if ( str[i] == delim ) {
            w[j] = 0;
            words.push_back( std::string( w ) );
            j = 0;
        } else if ( j == WORD_SIZE - 1 ) {
            std::cout << "word size exceeds maxium limit!" << std::endl;
            exit(1);
        } else {
            w[j++] = str[i];
        }
    }
}


// read data from csv file and convert it into sparsed data format
void read_data( const char * file, void ** data ) {
	fprintf( stdout, "============== READ DATA ===============\n" );
    // last stop date
    std::ifstream strm;
    int i, j, k;
    char str[LINE_WIDTH];
    std::vector< std::string > names;
    std::vector< std::string > values;
	std::string _s;
    float * mat = new float[FEATURE_DIM * LINES_TO_READ];
    float v;

    *data = mat;

    strm.open( file );
    if ( !strm.is_open() ) {
        std::cout << "failed to open data file!" << std::endl;
        exit(1);
    }

    // read header
    strm.getline( str, sizeof(str)-1 );
    split_line( names, str, ',' );
	fprintf( stdout, "CSV DATA HEAD READ!\n" );
	fprintf( stdout, "NAMES FOUDN: %lu.\n", names.size() );

	// for the last name: '\r' may be mixed in.
	_s = names[names.size()-1];
	i = _s.length();
	if ( _s[i-1] == '\r' ) {
		names[names.size()-1] = _s.substr(0, i-1 );
	}

    // read body
    for ( j=0; j<LINES_TO_READ; j++ ) {
        strm.getline( str, sizeof(str)-1 );
        split_line( values, str, ',' );
        for ( i=0, k=0; i<values.size(); i++ ) {
            k++;
            if ( names[i] == "LAST_STOP_DATE" ) {
                v = atof( values[i].c_str() );
                if ( v <= 0 || v > 12 ) {
                    mat[j * FEATURE_DIM + k - 1 ] = 13;
                }
                mat[j * FEATURE_DIM + k - 1 ] = v;

            } else if ( names[i] == "INNET_MONTHS" ) {

                v = atof( values[i].c_str() );

                if ( v <=0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 148 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(149);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "TOTAL_FLUX" ) {

                v = atof( values[i].c_str() );

                if ( v <=0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 1373 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(1374);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "LOCAL_FLUX_ZB" ) {

                v = atof( values[i].c_str() );
                mat[j * FEATURE_DIM + k - 1] = v;

            } else if ( names[i] == "JF_TIMES" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 1502 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(1503);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "NOROAM_LONG_JF_TIMES" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 386 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(387);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "ROAM_ZB" ) {

                v = atof( values[i].c_str() );
                mat[j * FEATURE_DIM + k - 1] = v;

            } else if ( names[i] == "ZHUJIAO_ZB" ) {

                v = atof( values[i].c_str() );
                mat[j * FEATURE_DIM + k - 1] = v;

            } else if ( names[i] == "TOLL_NUMS_ZB" ) {

                v = atof( values[i].c_str() );
                mat[j * FEATURE_DIM + k - 1] = v;

            } else if ( names[i] == "ACCT_FEE" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 170 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(171);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "ROAM_VOICE_FEE" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 49 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(50);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "ZENGZHI_FEE" ) {

                v = atof( values[i].c_str() );

                if ( v < 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 4 ) {
                    mat[j * FEATURE_DIM + k - 1] = 5;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = v;
                }

            } else if ( names[i] == "OWE_FEE" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 125 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(126);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "FLUX_TIME" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 47292 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(47293);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "YQ_OWE_MONTHS" ) {

                v = atof( values[i].c_str() );

                if ( v < 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 1;
                } else if ( v > 6 ) {
                    mat[j * FEATURE_DIM + k - 1] = 7;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = v;
                }

            } else if ( names[i] == "VAR_CDR_NUM" ) {

                v = atof( values[i].c_str() );
                mat[j * FEATURE_DIM + k - 1] = v;

            } else if ( names[i] == "CALL_DAYS" ) {

                v = atof( values[i].c_str() );
                mat[j * FEATURE_DIM + k - 1] = v;

            } else if ( names[i] == "LAST_CALL_TIME" ) {

                v = atof( values[i].c_str() );

                if ( v < 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 18 ) {
                    mat[j * FEATURE_DIM + k - 1] = 19;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = v;
                }

            } else if ( names[i] == "CALL_DURA_LAST7_CN" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 437 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf(438);
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "CELLID_NUM" ) {

                v = atof( values[i].c_str() );

                if ( v <= 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v > 86 ) {
                    mat[j * FEATURE_DIM + k - 1] = logf( 87 );
                } else {
                    mat[j * FEATURE_DIM + k - 1] = logf( v );
                }

            } else if ( names[i] == "PAY_MODE" ) {

                v = atoi ( values[i].c_str() );

                if ( v == 1 ) {
                    mat[j * FEATURE_DIM + k - 1] = 1;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v == 2) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 1;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                }

            } else if ( names[i] == "IS_GRP_MBR" ) {

                v = atoi ( values[i].c_str() );

                if ( v == 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 1;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                }

            } else if ( names[i] == "IS_TERM_IPHONE" ) {

                v = atoi ( values[i].c_str() );

                if ( v == 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 1;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                }

            } else if ( names[i] == "IS_USE_SMART" ) {

                v = atoi ( values[i].c_str() );

                if ( v == 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 1;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                }

            } else if ( names[i] == "NET_TYPE" ) {

                v = atoi ( values[i].c_str() );

                if ( v == 0 ) {
                    mat[j * FEATURE_DIM + k - 1] = 1;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v == 1 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 1;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v == 2 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 1;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                } else if ( v == 3 ) {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 1;
                } else {
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                    k++;
                    mat[j * FEATURE_DIM + k - 1] = 0;
                }

            } else {
                k--;
                // fill in flags
                if ( names[i] == "STABLE_FLAG" ) {
                    v = atoi( values[i].c_str() );
                    mat[j * FEATURE_DIM + FEATURE_DIM - 1] = v;
                }
            }
        }
    }
}

void clear_data( void ** data ) {
	fprintf( stdout, "============== CLEAN DATA ===============\n" );
    delete (float*)(*data);
    *data = NULL;
}


void test_model( void * data, float * model ) {
	fprintf( stdout, "============== TEST MODEL ===============\n" );
	int i, j;
	float s, t, accuracy, precision, recall, f_value;
	int true_accept = 0;
	int false_accept = 0;
	int true_refuse = 0;
	int false_refuse = 0;

	float * mat = (float*)data;
	for ( i=0; i<TEST_RECORD_NUM; i++ ) {
		s = 0.0;
		for ( j=0; j<FEATURE_DIM - 1; j++ ) {
			s += model[j] * mat[ (TRAIN_RECORD_NUM + i) * FEATURE_DIM + j ];
		}
		s += model[FEATURE_DIM - 1];
		s = F_SIGMOID( s );
		s = s > 0.5;
		t = mat[(TRAIN_RECORD_NUM + i) * FEATURE_DIM + FEATURE_DIM - 1];
		if ( t == 1 && s == 1 ) {
			true_accept ++;
		} else if ( t == 1 && s == 0 ) {
			false_refuse ++;
		} else if ( t == 0 && s == 1 ) {
			false_accept ++;
		} else if ( t == 0 && s == 0 ) {
			true_refuse ++;
		}
	}
	// print out model analysis
	accuracy = 1.0 * (true_accept + true_refuse) / TEST_RECORD_NUM;
	precision = 1.0 * true_accept/(true_accept + false_accept);
	recall = 1.0 * true_accept/(true_accept + false_refuse);
	f_value = precision * recall * 2.0 / ( precision + recall );
	fprintf( stdout, "accuracy: %.3f.\n", accuracy );
	fprintf( stdout, "precision: %.3f.\n", precision );
	fprintf( stdout, "recall: %.3f.\n", recall );
	fprintf( stdout, "F-value: %.3f.\n", f_value );
}


int main() {
#ifdef _WIN32
	LARGE_INTEGER begin;
	LARGE_INTEGER end;
	LARGE_INTEGER freq;
	QueryPerformanceFrequency( &freq );
	QueryPerformanceCounter( &begin );
#endif
    // read data
    void * data;
	float * model = new float[FEATURE_DIM];
    read_data( "_2G_FILTERED.CSV", &data );
    // training
    // before training, set random seeds
    srand( (unsigned long) 127 );
	cuda_lr( data, model );
#ifdef _WIN32
	QueryPerformanceCounter(&end);
	double millsec = 1000.0 * (end.QuadPart - begin.QuadPart)/freq.QuadPart;
	std::cout<<"cost time: "<< millsec << " milliseconds." << std::endl;
	fflush(stdout);
#endif
    // test model
	test_model( data, model );
    // clear
    clear_data( &data );
	delete [] model;

    return 0;
}
