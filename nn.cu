#include "hip/hip_runtime.h"
// nn.cpp : �������̨Ӧ�ó������ڵ㡣
//

#include <string.h>

#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <vector>
#include <utility>
#include <cmath>
#include <ctime>
#ifdef _WIN32_
#include "windows.h"
#endif

#include <hip/hip_runtime.h>
#include <>
#include "sm_20_atomic_functions.h"

using namespace std;


#define BLOCKSIZE 32


#define CUDA_CALL(func)\
  {\
    hipError_t e = (func);\
    if(e != hipSuccess)\
	cout << "LINE#"<<__LINE__<<": " << hipGetErrorString(e) << endl;\
  }

struct  NN_MODEL{
	int Train_num;
	int Test_num;
	int Train_count;
	int batch_size;
	int In_nodes;
	int Hiden_nodes;
	int Out_nodes;
	float learn_r;
	float *W1;
	float *W2;
	float *B1;
	float *B2;
};




//*********************************************** GPU function ***********************************//

/**
* ���ܣ�GPU���� C = sigmod( A��B + B_D )
* ���룺dev_A A����
* ���룺dev_B B����
* �����dev_C C����
* ���룺A_height A�ĸ߶�
* ���룺A_width A�Ŀ��
* ���룺B_height B�ĸ߶�
* ���룺B_width B�Ŀ��
* ���룺B_D ƫ������
*/
__global__ void BP_Calculate_mmul(float *dev_A, float *dev_B, float *dev_C, int A_height, int A_width, int B_height, int B_width, float* B_D)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;
	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < A_width; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < A_width) && (y_id < A_height))
			As[threadIdx.y][threadIdx.x] = dev_A[y_id * A_width + colA]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < B_width) && (rowB < B_height))
			Bs[threadIdx.y][threadIdx.x] = dev_B[rowB * B_width + x_id]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}

	if (x_id < B_width && y_id < A_height)
	{
		dev_C[y_id*B_width + x_id] = 1 / ((1 + exp(-(Cvalue + B_D[x_id]))));
		//gpu_sigmod(dev_C, Cvalue + B_D[x_id], y_id*B_width + x_id);
	}
}




/**
* ���ܣ�GPU���� �������޸��� C = A *(1-A) *(B-A)
* ���룺dev_A ���������
* ���룺dev_B ��ǩ����
* �����dev_C �������޸�������
* ���룺height �߶�
* ���룺width ���
*/
__global__ void BP_Calculate_out_update(float *dev_A, float *dev_B, float *dev_C, int height, int width)
{
	int row = blockDim.y * blockIdx.y + threadIdx.y;//������
	int col = blockDim.x * blockIdx.x + threadIdx.x; //������

	if (row < height && col < width)
	{
		dev_C[row*width + col] = dev_A[row*width + col] * (1 - dev_A[row*width + col])*(dev_B[row*width + col] - dev_A[row*width + col]);
	}
}




/**
* ���ܣ�GPU���� ���ز���޸��� D =  (A��B') * C *(1-C)
* ���룺dev_A �������޸�������
* ���룺dev_B ���ز㵽������Ȩֵ����
* ���룺dev_C ���ز�����
* �����dev_D ���ز���޸�������
* ���룺A_height A�ĸ߶�
* ���룺A_width A�Ŀ��
* ���룺B_height B�ĸ߶�
* ���룺B_width B�Ŀ��
*/
__global__ void BP_Calculate_hiden_update(float *dev_A, float *dev_B, float *dev_C, float *dev_D, int A_height, int A_width, int B_height, int B_width)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < A_width; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < A_width) && (y_id < A_height))
			As[threadIdx.y][threadIdx.x] = dev_A[y_id * A_width + colA]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < B_height) && (rowB <B_width))
			Bs[threadIdx.y][threadIdx.x] = dev_B[x_id * B_width + rowB]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}

	if (x_id < B_height && y_id < A_height)
	{
		//ChgH[batch_size][Hiden_nodes] = temp[batch_size][Hiden_nodes] .* O1[batch_size][Hiden_nodes] .*(1-O1[batch_size][Hiden_nodes])
		dev_D[y_id * B_height + x_id] = Cvalue*dev_C[y_id * B_height + x_id] * (1 - dev_C[y_id * B_height + x_id]);
	}
}




/**
* ���ܣ�GPU����Ȩ����  C = C + lr *(A'��B);
* ���룺dev_A ��һ������
* ���룺dev_B �ò���޸���
* ���룺dev_C ��һ�㵽�ò��Ȩ����
* �����dev_C ��һ�㵽�ò��Ȩ����
* ���룺A_height A�ĸ߶�
* ���룺A_width A�Ŀ��
* ���룺B_height B�ĸ߶�
* ���룺B_width B�Ŀ��
*/
__global__ void BP_Calculate_W_update(float *dev_A, float *dev_B, float *dev_C, int A_height, int A_width, int B_height, int B_width, float learn_r)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < A_height; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < A_height) && (y_id < A_width))
			As[threadIdx.y][threadIdx.x] = dev_A[colA * A_width + y_id]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;
		if ((x_id < B_width) && (rowB < B_height))
			Bs[threadIdx.y][threadIdx.x] = dev_B[rowB * B_width + x_id]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}

	if (x_id < B_width && y_id < A_width)
	{
		dev_C[y_id * B_width + x_id] += Cvalue * learn_r;
	}
}





/**
* ���ܣ�GPU����ƫ��
* ���룺dev_A �ò���޸���
* ���룺dev_B ƫ������
* ���룺A_height A�ĸ߶�
* ���룺A_width A�Ŀ��
*/
__global__ void BP_Calculate_B_update(float *A, float*B, int A_height, int A_width, float learn_r)
{
	int col = threadIdx.x; //��

	float sum = 0.0;
	for (int i = 0; i < A_height; i++)
	{
		sum += A[i*A_width+col];
	}

	if (col < A_width)
	{
		B[col] = sum * learn_r;
	}
}

//***********************************GPU function end ***************************************//



//read data
void read_data(int data_num, int in_nodes, int out_nodes, float *data_x, float *data_y, char *data_x_file, char *data_y_file)
{
	FILE *fp1, *fp2;
	if ((fp1 = fopen(data_x_file, "r")) == NULL){
		printf("can not open the in file\n");
		exit(0);
	}
	for (int i = 0; i < data_num; i++)
	{
		for (int j = 0; j < in_nodes; j++)
		{
			fscanf(fp1, "%f", &data_x[i*in_nodes + j]);
		}
	}
	fclose(fp1);

	if ((fp2 = fopen(data_y_file, "r")) == NULL){
		printf("can not open the out file\n");
		exit(0);
	}
	for (int i = 0; i < data_num; i++)
	{
		for (int j = 0; j < out_nodes; j++)
		{
			fscanf(fp2, "%f", &data_y[i*out_nodes + j]);
		}
	}
	fclose(fp2);

	printf("read data end.\n");

}

//init class model
void init_class_model(NN_MODEL * model)
{
	model->Train_num = 1000000;
	model->Test_num = 10000;
	model->Train_count = 5;
	model->batch_size = 100;
	model->In_nodes = 100;
	model->Hiden_nodes = 50;
	model->Out_nodes = 2;
	model->learn_r = 0.1;

	//malloc
	model->W1 = (float*)malloc(model->In_nodes * model->Hiden_nodes * sizeof(float));
	model->W2 = (float*)malloc(model->Hiden_nodes * model->Out_nodes * sizeof(float));
	model->B1 = (float*)malloc(model->Hiden_nodes * sizeof(float));
	model->B2 = (float*)malloc(model->Out_nodes * sizeof(float));


	//init
	srand((unsigned)time(NULL));
	for (int i = 0; i < model->In_nodes; i++)
	{
		for (int j = 0; j < model->Hiden_nodes; j++)
		{
			model->W1[i*model->Hiden_nodes + j] = (rand()*2.0 / RAND_MAX - 1) / 2.0;
		}
	}
	for (int i = 0; i < model->Hiden_nodes; i++)
	{
		model->B1[i] = 1.0;
	}

	for (int i = 0; i < model->Hiden_nodes; i++)
	{
		for (int j = 0; j < model->Out_nodes; j++)
		{
			model->W2[i*model->Out_nodes + j] = (rand()*2.0 / RAND_MAX - 1) / 2.0;
		}
	}
	for (int i = 0; i < model->Out_nodes; i++)
	{
		model->B2[i] = 1.0;
	}

	printf("class model init end.\n");

}

//class model trian
void class_model_train(NN_MODEL *model, float *train_x, float *train_y)
{
	printf("feature extracted model training ......\n");



	int dev_num;
	hipGetDeviceCount(&dev_num);
	if ( dev_num < 1 ) {
		fprintf(stdout, "NO GPU AVAILABLE FOR COMUPTING!\n");
		exit(1);
	}
	fprintf( stdout, "GPU NUM: %d\n", dev_num );
	srand(time(NULL));
	CUDA_CALL(hipSetDevice(1));


	///* �����̸߳���߳̿� */
	dim3 dimBlock2D(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid2D_batch_in_hiden((model->Hiden_nodes + BLOCKSIZE - 1) / dimBlock2D.x, (model->batch_size + BLOCKSIZE - 1) / dimBlock2D.y); //�������
	dim3 dimGrid2D_batch_hiden_out((model->Out_nodes + BLOCKSIZE - 1) / dimBlock2D.x, (model->batch_size + BLOCKSIZE - 1) / dimBlock2D.y); //�������
	dim3 dimGrid2D_in_hiden((model->Hiden_nodes + BLOCKSIZE - 1) / dimBlock2D.x, (model->In_nodes + BLOCKSIZE - 1) / dimBlock2D.y); //�������
	dim3 dimGrid2D_hiden_out((model->Out_nodes + BLOCKSIZE - 1) / dimBlock2D.x, (model->Hiden_nodes + BLOCKSIZE - 1) / dimBlock2D.y); //�������


	//**************�����豸�˿ռ�*********************
	int Train_num = model->Train_num;
	//����
	float *train_x_D, *train_y_D;
	hipMalloc((void**)&train_x_D, Train_num*model->In_nodes * sizeof(float));
	hipMalloc((void**)&train_y_D, Train_num*model->Out_nodes * sizeof(float));
	//Ȩֵ
	float *W1_D, *W2_D;
	hipMalloc((void**)&W1_D, model->In_nodes*model->Hiden_nodes * sizeof(float));
	hipMalloc((void**)&W2_D, model->Hiden_nodes *model->Out_nodes* sizeof(float));
	//ƫ��
	float *B1_D, *B2_D;
	hipMalloc((void**)&B1_D, model->Hiden_nodes * sizeof(float));
	hipMalloc((void**)&B2_D, model->Out_nodes * sizeof(float));

	//���ز�����������
	float * O1_D, *O2_D;
	hipMalloc((void**)&O1_D, model->batch_size * model->Hiden_nodes * sizeof(float));
	hipMalloc((void**)&O2_D, model->batch_size * model->Out_nodes * sizeof(float));
	//���������ز��Ȩֵ�޸���
	float * ChgO_D, *ChgH_D;
	hipMalloc((void**)&ChgO_D, model->batch_size * model->Out_nodes * sizeof(float));
	hipMalloc((void**)&ChgH_D, model->batch_size * model->Hiden_nodes * sizeof(float));

	printf("device memory malloc end!\n");

	//**************�������豸����*********************
	hipMemcpy(train_x_D, train_x, Train_num*model->In_nodes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(train_y_D, train_y, Train_num*model->Out_nodes * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(W1_D, model->W1, model->In_nodes*model->Hiden_nodes*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(W2_D, model->W2, model->Hiden_nodes *model->Out_nodes*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B1_D, model->B1, model->Hiden_nodes*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B2_D, model->B2, model->Out_nodes*sizeof(float), hipMemcpyHostToDevice);

	printf("copy data from CPU to GPU end!\n");

	/* ��¼ʱ�� */
	hipEvent_t start_GPU_t, end_GPU_t;
	float elaspsedTime_t;
	hipEventCreate(&start_GPU_t);
	hipEventCreate(&end_GPU_t);
	hipEventRecord(start_GPU_t, 0);


	//ѵ�����
	for (int t_c = 0; t_c < model->Train_count; t_c++)
	{
		
		/* ��¼ʱ�� */
		hipEvent_t start_GPU, end_GPU;
		float elaspsedTime;
		hipEventCreate(&start_GPU);
		hipEventCreate(&end_GPU);
		hipEventRecord(start_GPU, 0);

		//ÿ��ѵ������
		for (int index = 0; index < (Train_num - model->batch_size); index += model->batch_size)
		{
			//���ز����
			//O1[batch_size][Hiden_nodes] = S (train_X[model->batch_size][In_nodes] �� W1[In_nodes][Hiden_nodes] + B1_D[Hiden_nodes])
			BP_Calculate_mmul << <dimGrid2D_batch_in_hiden, dimBlock2D >> >(&train_x_D[index*model->In_nodes], W1_D, O1_D, model->batch_size, model->In_nodes, model->In_nodes, model->Hiden_nodes, B1_D);

			//��������  
			//O2[batch_size][Out_nodes] = S (O1[batch_size][Hiden_nodes] �� W2[Hiden_nodes][Out_nodes] + B2_D[Out_nodes])
			BP_Calculate_mmul << <dimGrid2D_batch_hiden_out, dimBlock2D >> >(O1_D, W2_D, O2_D, model->batch_size, model->Hiden_nodes, model->Hiden_nodes, model->Out_nodes, B2_D);
			   

			//����������Ȩֵ�޸���
			//ChgO[batch_size][Out_nodes] = O2[batch_size][Out_nodes] * (1 - O2) * (train_Y[batch_size][Out_nodes] - O2);
			BP_Calculate_out_update << <dimGrid2D_batch_hiden_out, dimBlock2D >> >(O2_D, &train_y_D[index*model->Out_nodes], ChgO_D, model->batch_size, model->Out_nodes);


			//�������ز��Ȩ�޸���
			//ChgH[batch_size][Hiden_nodes] = (ChgO[batch_size][Out_nodes] �� W2[Hiden_nodes][Out_nodes]') * O1[batch_size][Hiden_nodes] *(1-O1)
			BP_Calculate_hiden_update << <dimGrid2D_batch_in_hiden, dimBlock2D >> >(ChgO_D, W2_D, O1_D, ChgH_D, model->batch_size, model->Out_nodes, model->Hiden_nodes, model->Out_nodes);

			//�޸������Ȩ����
			// W2[Hiden_nodes][Out_nodes] = W2[Hiden_nodes][Out_nodes] + learn_r * O1[batch_size][Hiden_nodes] * ChgO[batch_size][Out_nodes]
			//����ת��ΪW2[Hiden_nodes][Out_nodes] = learn_r * (O1[batch_size][Hiden_nodes]'�� ChgO[batch_size][Out_nodes]);
			BP_Calculate_W_update << <dimGrid2D_hiden_out, dimBlock2D >> >(O1_D, ChgO_D, W2_D, model->batch_size, model->Hiden_nodes, model->batch_size, model->Out_nodes,model->learn_r);

			//�޸����ز�Ȩ����
			//W1[In_nodes][Hiden_nodes] = W1[In_nodes][Hiden_nodes] + learn_r * train_X[batch_size][In_nodes] * ChgH[batch_size][Hiden_nodes]
			//W1[In_nodes][Hiden_nodes] = learn_r * (train_X[batch_size][In_nodes]' �� ChgH[batch_size][Hiden_nodes])
			BP_Calculate_W_update << <dimGrid2D_in_hiden, dimBlock2D >> >(&train_x_D[index*model->In_nodes], ChgH_D, W1_D, model->batch_size, model->In_nodes, model->batch_size, model->Hiden_nodes, model->learn_r);

			//�޸�������ƫ��
			//B2[Out_nodes] = B2[Out_nodes] + learn_r * ChgO[batch_size][Out_nodes]
			BP_Calculate_B_update << <1, model->Out_nodes >> >(ChgO_D, B2_D, model->batch_size, model->Out_nodes, model->learn_r);

			//�޸����ز��ƫ��
			//B1[Hiden_nodes] = B1[Hiden_nodes] + learn_r * ChgH[batch_size][Hiden_nodes]
			BP_Calculate_B_update << <1, model->Hiden_nodes >> >(ChgH_D, B1_D, model->batch_size, model->Hiden_nodes, model->learn_r);
			//ѵ��һ���ν���
		}

		/* ��ʱ���� */
		hipEventRecord(end_GPU, 0);
		hipEventSynchronize(end_GPU);
		hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

		
		printf("[%d/%d], time[ms] = %0.5f\n", t_c + 1, model->Train_count, elaspsedTime);
		
	}

	printf("training end!\n");


	/* ��ʱ���� */
	hipEventRecord(end_GPU_t, 0);
	hipEventSynchronize(end_GPU_t);
	hipEventElapsedTime(&elaspsedTime_t, start_GPU_t, end_GPU_t);
	printf("*******************************************\n");
	printf("total training time: %0.5f (ms).\n", elaspsedTime_t);
	printf("*******************************************\n");

	
	//�ͷ�
	hipFree(train_x_D);
	hipFree(train_y_D);
	hipFree(W1_D);
	hipFree(W2_D);
	hipFree(O1_D);
	hipFree(O2_D);
	hipFree(ChgO_D);
	hipFree(ChgH_D);

	printf("device memory free end��\n");



	printf("feature extracted model training end!\n");
}

int main()
{
	//model
	NN_MODEL *model = (NN_MODEL*)malloc(sizeof(struct NN_MODEL));
	init_class_model(model);

	//train data
	float *train_x = (float *)malloc(model->Train_num * model->In_nodes * sizeof(float));
	float *train_y = (float *)malloc(model->Train_num * model->Out_nodes * sizeof(float));
	read_data(model->Train_num, model->In_nodes, model->Out_nodes, train_x, train_y, "train_x.txt", "train_y.txt");

	//class model trian
	class_model_train(model, train_x, train_y);

	getchar();
	//free data
	free(train_x);
	free(train_y);

	free(model->W1);
	free(model->W2);
	free(model->B1);
	free(model->B2);
	free(model);

	getchar();

	return 0;
}

